#include "hip/hip_runtime.h"
/*
 * File:   main.cpp
 * Author: jjbillings
 *
 * Created on October 16, 2016, 9:09 PM
 */

#include<cstdlib>
#include<stdio.h>
#include<queue>
#include<stack>
#include<iostream>
#include<fstream>
#include<ctime>
#include"nets.h"

using namespace std;

#define NUM_CONNECTIONS 500
#define MAX_CHANNELS 30
#define SAMPLES 1

struct SimplePath;
struct Path;
struct Edge;
struct Connection;
struct Connection;
struct Channel;

struct Channel{
    bool primary; //is this channel used for a primary path?
    int numBackups; //total protected;
    Connection *backupsOnChannel[NUM_CONNECTIONS];//Realistically, there will be far fewer than NUM_CONNECTIONS
};

struct Edge {
    int edgeNum;
    int v1;
    int v2;
    int load; //load <= MAX_CHANNELS. Also, load is the sum of the primary AND backups paths using it.
    int totalProtected;
};

struct SimplePath {
    int sourceNode;
    int destNode;

    int hops;
    int index;

    Edge *edges[N_NODES];
};

struct Path {
    int sourceNode;
    int destNode;

    int hops;
    int index;
    int cost;

    //Every path that uses a particular edge just has a reference to it (not a copy), so they can each manipulate it.
    Edge *edges[N_NODES];
    bool freeEdges[N_NODES]; //whether or not that edge has a cost of 0
    int channelNum[N_NODES]; //Channel number for each edge that it uses
    bool primary;
    bool active;
};

struct Connection {
    int sourceNode;
    int destNode;
    int combinedCost;
    bool validBackup;
    bool validPrimary;
    Path *backupPath;
    Path *primaryPath;
};

void readGraphReorderEdgeList(int vertexList[],Edge compactEdgeList[2*N_EDGES],Edge reorderedEdgeList[2*N_NODES]);

int computeAllSimplePathsN(SimplePath **ps, int *vertexList, Edge *edgeList, int sourceNode, int destNode, int hops);
void simulate(int *vertexList, Edge *edgeList);
void simulate_GPU(int *vertexList, Edge *edgeList);
int determineCompatibleBackups(SimplePath *p, int *potPathInd, int numPossiblePaths, int pInd);
void computeCostForBackups(SimplePath *p, int *potPathInd, int numPotPaths, int backupIndex, int *pathCosts,Channel cs[2*N_EDGES][MAX_CHANNELS]);
void selectChannels(Connection *c, Channel chan[2*N_EDGES][MAX_CHANNELS]);
void increaseLoad(Connection *connection, Channel channels[2*N_EDGES][MAX_CHANNELS]);

int vertexList[N_NODES+1];
Edge edgeList[2*N_EDGES];
Edge reorderedEdgeList[2*N_EDGES];
Connection cons[NUM_CONNECTIONS];
Channel channels[2*N_EDGES][MAX_CHANNELS];

/*
 *TODO: I totally thought I made the algorithm be based on BFS, but it is in fact based on DFS.
 *So REVERSE the order of the edge list. Currently, the neighbor with the lowest degree gets pushed
 *to the "bottom" of the stack, so we end up computing the path with high-degree nodes in it...
 */
int main(int argc, char** argv) {
    cout <<"Welcome to main\n";

    for(int f = 0; f < (2*N_EDGES); ++f){
        for(int g = 0; g < MAX_CHANNELS; ++g) {
            channels[f][g].numBackups = 0;
            channels[f][g].primary = false;
        }
    }
    readGraphReorderEdgeList(vertexList,edgeList,reorderedEdgeList);

    srand(time(NULL));

    simulate(vertexList,edgeList);
    return 0;
}

void simulate_GPU(int *vertexList, Edge *edgeList){
    int connectionNum = 0;
    //We want to compute and store all possible paths between our source and desitination.
    SimplePath **ps = new SimplePath*[N_NODES * N_NODES]; //Storage for paths
    int *npaths = new int[N_NODES*N_NODES];

    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        ps[i] = new SimplePath[NUM_CONNECTIONS];
    }

    cout <<"ps created\n";

    //We COULD parallelize this by giving a thread a source/dest combo to compute the paths of. potentially beneficial for large graphs
    for(int src = 0; src < N_NODES; ++src) {
        for(int dest = 0; dest < N_NODES; ++dest) {
            if(src != dest) {
                int index = (src*N_NODES)+dest;
                npaths[index] = computeAllSimplePathsN(ps,vertexList,edgeList,src,dest,N_NODES);
                cout <<"All simple paths computed and stored! " << npaths[index] << " paths between " << src << " and " << dest << "\n";
            }
        }
    }
    //At this point, we COULD delete[] any paths in the array that we didn't use.
    cout << "all simple paths computed!\n";


    //Attempt to allocate SOME connection onto the network
    //int s = 0;
    //int d = 9;
    int s = rand() % N_NODES;
    int d = rand() % N_NODES;
    while(s == d) {
        s = rand()%N_NODES;
        d = rand()%N_NODES;
    }

    //Allocate storage for the potential primary/backup path combos
    int index = (s*N_NODES) + d;
    int numPossiblePaths = npaths[index];

    //Stores indices into the ps[index][] array for each disjoint backup path.
    //potPathInd[i][j] = k where ps[index][k] is a path that is edge-disjoint from ps[index][i].
    int ** potPathInd = new int*[numPossiblePaths];
    for(int i = 0; i < numPossiblePaths; ++i) {
        potPathInd[i] = new int[numPossiblePaths];
    }


    //--------------Find all paths which are edge-disjoint from this primary--------------//
    int k = -1;
    //On the GPU, instead of iterating i..numPossiblePaths, we would give thread_i backup_i
    for(int i = 0; i < numPossiblePaths; ++i) {
        k = determineCompatibleBackups(ps[index],potPathInd[i],numPossiblePaths,i);
        //cout << "Number of paths which are disjoint from this primary path: " << k << "\n";
    }



    //--------------Compute Cost for each backup path--------------//
    int ** pathCosts = new int*[numPossiblePaths];
    for(int i = 0; i < numPossiblePaths; ++i) {
        pathCosts[i] = new int[numPossiblePaths];
    }

    for(int i = 0; i < numPossiblePaths; ++i) {
        computeCostForBackups(ps[index],potPathInd[i],numPossiblePaths,i,pathCosts[i],channels);
    }



    //--------------Select cheapest connection--------------//
    int minCost = 100000000;
    int minPrimInd = -1;
    int minBackInd = -1;

    for(int p = 0; p < numPossiblePaths; ++p) {
        int backInd = 0;
        int primaryCost = ps[index][p].hops;

        while(pathCosts[p][backInd] != -1) {
            if((pathCosts[p][backInd] + primaryCost) < minCost) {
                minCost = (pathCosts[p][backInd] + primaryCost);
                minPrimInd = p;
                minBackInd = backInd;
            }
            backInd++;
        }
    }
    cout << "Min cost is: " << minCost << "\n";



    //--------------Store the connection--------------//
    cons[connectionNum].sourceNode = s;
    cons[connectionNum].destNode = d;
    cons[connectionNum].combinedCost = minCost;
    cons[connectionNum].validBackup = true;
    cons[connectionNum].validPrimary = true;
    cons[connectionNum].backupPath = new Path();
    cons[connectionNum].primaryPath = new Path();
    (*cons[connectionNum].primaryPath).hops = ps[index][minPrimInd].hops;
    (*cons[connectionNum].primaryPath).index = ps[index][minPrimInd].index;
    (*cons[connectionNum].primaryPath).primary = true;
    (*cons[connectionNum].backupPath).hops = ps[index][potPathInd[minPrimInd][minBackInd]].hops;
    (*cons[connectionNum].backupPath).index = ps[index][potPathInd[minPrimInd][minBackInd]].index;

    for(int p = 0; p <= ps[index][minPrimInd].index; ++p) {
        (*cons[connectionNum].primaryPath).edges[p] = ps[index][minPrimInd].edges[p];
        (*cons[connectionNum].primaryPath).freeEdges[p] = false;
    }
    for(int p = 0; p <= ps[index][potPathInd[minPrimInd][minBackInd]].index; ++p) {
        (*cons[connectionNum].backupPath).edges[p] = ps[index][potPathInd[minPrimInd][minBackInd]].edges[p];
    }

    //Select Channels
    selectChannels(&cons[connectionNum],channels);

    //Increase the network load
    increaseLoad(&cons[connectionNum],channels);


    //--------------Print Network Load--------------//
    for(int m = 0; m < 2*N_EDGES; ++m) {
        cout << "LOAD: " << edgeList[m].v1 << " -> " << edgeList[m].v2 << ": " << edgeList[m].load << " | TP: " << edgeList[m].totalProtected << " | ";
        if(edgeList[m].load > 0) {
            for(int c = 0; c < edgeList[m].load; ++c) {
                cout << "C" << c << ": " << channels[m][c].numBackups << " ";
                if(channels[m][c].primary == true) {
                    cout << "P ";
                }
            }
        }
        cout << "\n";

    }


    //--------------Clean up memory--------------//
    for(int i = 0; i < numPossiblePaths; ++i) {
        delete[] potPathInd[i];
    }
    delete[] potPathInd;

    for(int i = 0; i < numPossiblePaths; ++i) {
        delete[] pathCosts[i];
    }
    delete[] pathCosts;
    connectionNum++;

    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        delete[] ps[i];
    }
    delete[] ps;
    delete[] npaths;
    cout << "ps and npaths deleted\n";
}


void simulate(int *vertexList, Edge *edgeList){
    int connectionNum = 0;
    //We want to compute and store all possible paths between our source and desitination.
    SimplePath **ps = new SimplePath*[N_NODES * N_NODES]; //Storage for paths
    int *npaths = new int[N_NODES*N_NODES];

    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        ps[i] = new SimplePath[NUM_CONNECTIONS];
    }

    cout <<"ps created\n";

    //We COULD parallelize this by giving a thread a source/dest combo to compute the paths of. potentially beneficial for large graphs
    for(int src = 0; src < N_NODES; ++src) {
        for(int dest = 0; dest < N_NODES; ++dest) {
            if(src != dest) {
                int index = (src*N_NODES)+dest;
                npaths[index] = computeAllSimplePathsN(ps,vertexList,edgeList,src,dest,N_NODES);
                cout <<"All simple paths computed and stored! " << npaths[index] << " paths between " << src << " and " << dest << "\n";
            }
        }
    }
    //At this point, we COULD delete[] any paths in the array that we didn't use.
    cout << "all simple paths computed!\n";


    for(int num = 0; num < 45; ++num) {
    //Attempt to allocate SOME connection onto the network
    //int s = 0;
    //int d = 9;
    int s = rand() % N_NODES;
    int d = rand() % N_NODES;
    while(s == d) {
        s = rand()%N_NODES;
        d = rand()%N_NODES;
    }

    //Allocate storage for the potential primary/backup path combos
    int index = (s*N_NODES) + d;
    int numPossiblePaths = npaths[index];

    //Stores indices into the ps[index][] array for each disjoint backup path.
    //potPathInd[i][j] = k where ps[index][k] is a path that is edge-disjoint from ps[index][i].
    int ** potPathInd = new int*[numPossiblePaths];
    for(int i = 0; i < numPossiblePaths; ++i) {
        potPathInd[i] = new int[numPossiblePaths];
    }


    //--------------Find all paths which are edge-disjoint from this primary--------------//
    int k = -1;
    //On the GPU, instead of iterating i..numPossiblePaths, we would give thread_i backup_i
    for(int i = 0; i < numPossiblePaths; ++i) {
        k = determineCompatibleBackups(ps[index],potPathInd[i],numPossiblePaths,i);
        //cout << "Number of paths which are disjoint from this primary path: " << k << "\n";
    }



    //--------------Compute Cost for each backup path--------------//
    int ** pathCosts = new int*[numPossiblePaths];
    for(int i = 0; i < numPossiblePaths; ++i) {
        pathCosts[i] = new int[numPossiblePaths];
    }

    for(int i = 0; i < numPossiblePaths; ++i) {
        computeCostForBackups(ps[index],potPathInd[i],numPossiblePaths,i,pathCosts[i],channels);
    }



    //--------------Select cheapest connection--------------//
    int minCost = 100000000;
    int minPrimInd = -1;
    int minBackInd = -1;

    for(int p = 0; p < numPossiblePaths; ++p) {
        int backInd = 0;
        int primaryCost = ps[index][p].hops;

        while(pathCosts[p][backInd] != -1) {
            if((pathCosts[p][backInd] + primaryCost) < minCost) {
                minCost = (pathCosts[p][backInd] + primaryCost);
                minPrimInd = p;
                minBackInd = backInd;
            }
            backInd++;
        }
    }
    cout << "Min cost is: " << minCost << "\n";



    //--------------Store the connection--------------//
    cons[connectionNum].sourceNode = s;
    cons[connectionNum].destNode = d;
    cons[connectionNum].combinedCost = minCost;
    cons[connectionNum].validBackup = true;
    cons[connectionNum].validPrimary = true;
    cons[connectionNum].backupPath = new Path();
    cons[connectionNum].primaryPath = new Path();
    (*cons[connectionNum].primaryPath).hops = ps[index][minPrimInd].hops;
    (*cons[connectionNum].primaryPath).index = ps[index][minPrimInd].index;
    (*cons[connectionNum].primaryPath).primary = true;
    (*cons[connectionNum].backupPath).hops = ps[index][potPathInd[minPrimInd][minBackInd]].hops;
    (*cons[connectionNum].backupPath).index = ps[index][potPathInd[minPrimInd][minBackInd]].index;

    for(int p = 0; p <= ps[index][minPrimInd].index; ++p) {
        (*cons[connectionNum].primaryPath).edges[p] = ps[index][minPrimInd].edges[p];
        (*cons[connectionNum].primaryPath).freeEdges[p] = false;
    }
    for(int p = 0; p <= ps[index][potPathInd[minPrimInd][minBackInd]].index; ++p) {
        (*cons[connectionNum].backupPath).edges[p] = ps[index][potPathInd[minPrimInd][minBackInd]].edges[p];
    }

    //Select Channels
    selectChannels(&cons[connectionNum],channels);

    //Increase the network load
    increaseLoad(&cons[connectionNum],channels);


    //--------------Print Network Load--------------//
    for(int m = 0; m < 2*N_EDGES; ++m) {
        cout << "LOAD: " << edgeList[m].v1 << " -> " << edgeList[m].v2 << ": " << edgeList[m].load << " | TP: " << edgeList[m].totalProtected << " | ";
        if(edgeList[m].load > 0) {
            for(int c = 0; c < edgeList[m].load; ++c) {
                cout << "C" << c << ": " << channels[m][c].numBackups << " ";
                if(channels[m][c].primary == true) {
                    cout << "P ";
                }
            }
        }
        cout << "\n";

    }


    //--------------Clean up memory--------------//
    for(int i = 0; i < numPossiblePaths; ++i) {
        delete[] potPathInd[i];
    }
    delete[] potPathInd;

    for(int i = 0; i < numPossiblePaths; ++i) {
        delete[] pathCosts[i];
    }
    delete[] pathCosts;
    connectionNum++;
}//end loop

    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        delete[] ps[i];
    }
    delete[] ps;
    delete[] npaths;
    cout << "ps and npaths deleted\n";
}

void increaseLoad(Connection *connection, Channel channels[2*N_EDGES][MAX_CHANNELS]) {
    if((*(*connection).primaryPath).index < 0) {
        cout << "Primary Path DNE?\n";
        return;
    }
    //Increment the network load; put the backup on the channels

    //Here we are incrementing the network load for the PRIMARY PATH
    for(int i = 0; i <= (*(*connection).primaryPath).index; ++i) {

        //Every edge in the primary path gets its load increased
        channels[(*(*(*connection).primaryPath).edges[i]).edgeNum][(*(*connection).primaryPath).channelNum[i]].primary = true;
        channels[(*(*(*connection).primaryPath).edges[i]).edgeNum][(*(*connection).primaryPath).channelNum[i]].backupsOnChannel[0] = connection;
        channels[(*(*(*connection).primaryPath).edges[i]).edgeNum][(*(*connection).primaryPath).channelNum[i]].numBackups += 1;
        (*(*(*connection).primaryPath).edges[i]).load += 1;
        (*(*(*connection).primaryPath).edges[i]).totalProtected += 1;
    }

    //Here we are increasing the network load for the BACKUP PATH
    for(int i = 0; i <= (*(*connection).backupPath).index; ++i) {
        //Temp
        Edge *e = (*(*connection).backupPath).edges[i];
        int cNum = (*(*connection).backupPath).channelNum[i];

        //first path to use this channel, or this is not a free edge for the backup path.
        //if(channels[(*e).edgeNum][cNum].numBackups == 0 || (*(*connection).backupPath).freeEdges[i] == false) {
        if((*(*connection).backupPath).freeEdges[i] == false) {
            (*e).load += 1;
        }

        //Marks that the connection is protected on this channel.
        int en = (*e).edgeNum;
        int numbs = channels[en][cNum].numBackups;
        channels[en][cNum].primary = false;
        channels[en][cNum].backupsOnChannel[numbs] = connection;
        channels[en][cNum].numBackups += 1;
        (*e).totalProtected +=1;
    }

}

//TODO: This method contains a lot of redundant code that is also in computeCostForBackups. Consider combining.
//I wanted to modularize the code as much as possible this time around, which is why there's so much redundancy in this method.
void selectChannels(Connection *c, Channel chan[2*N_EDGES][MAX_CHANNELS]) {

    cout << "prim\n";
    for(int i = 0; i <= (*(*c).primaryPath).index; ++i) {
        cout << (*(*(*c).primaryPath).edges[i]).v1 << " -> " << (*(*(*c).primaryPath).edges[i]).v2 << "\n";
    }
    cout << "back\n";
    for(int i = 0; i <= (*(*c).backupPath).index; ++i) {
        cout << (*(*(*c).backupPath).edges[i]).v1 << " -> " << (*(*(*c).backupPath).edges[i]).v2 << "\n";
    }

    int edgeNum = -1;
    //Select Primary path channels;
    for(int p = 0; p <= (*(*c).primaryPath).index; ++p){
        edgeNum = (*(*(*c).primaryPath).edges[p]).edgeNum;
        bool allSet = false;
        for(int ch = 0; !allSet && ch < MAX_CHANNELS; ++ch) {
            if(chan[edgeNum][ch].numBackups == 0) {
                allSet = true;
                (*(*c).primaryPath).channelNum[p] = ch;
            }
        }
    }

    for(int e = 0; e <= (*(*c).backupPath).index; ++e) {
        bool free = false;
        edgeNum = (*(*(*c).backupPath).edges[e]).edgeNum;
        int firstOpenChannel = MAX_CHANNELS+1;

        for(int ch = 0; !free && ch < MAX_CHANNELS; ++ch) {

            if(chan[edgeNum][ch].primary == true) {
                continue;
            }

            //At this point, we know that there are no primary paths on this channel
            //Thus we must check and see if it is "free".

            //we COULD use this channel, but there may be a "free" one further down.
            if(chan[edgeNum][ch].numBackups == 0) {
                if(ch < firstOpenChannel) {
                    firstOpenChannel = ch;
                }
                continue;
            }

            bool disjoint = true;

            //Check every connection currently on protected on the channel
            for(int bup = 0; disjoint && bup < chan[edgeNum][ch].numBackups; ++bup) {

                //At this point, we know that there is at least one path protected on this channel.
                //Technically, we should also know that it's not a primary path.

                //for each edge of the protected connection's primary path
                for(int e2 = 0; disjoint && e2 <= (*(*chan[edgeNum][ch].backupsOnChannel[bup]).primaryPath).index; ++e2) {

                    //see if its the same edge as used by our primary path.
                    for(int e3 = 0; disjoint && e3 <= (*(*c).primaryPath).index; ++e3 ) {

                        if((*(*chan[edgeNum][ch].backupsOnChannel[bup]).primaryPath).edges[e2] == (*(*c).primaryPath).edges[e3]) {
                            //There is a non-disjoint primary path on this channel, so it is unusable.
                            //goto CHANNEL_LOOP_END;
                            disjoint = false;
                        }
                    }
                }
            }

            if(disjoint) {
                //This channel is free
                free = true;
                (*(*c).backupPath).channelNum[e] = ch;
                (*(*c).backupPath).freeEdges[e] = true;
            }
        }

        if((*(*c).backupPath).freeEdges[e] == false) {
            (*(*c).backupPath).channelNum[e] = firstOpenChannel;
        }
    }
    cout << "all set?\n";
}

//TODO: Need to test once we actually start loading the network.
void computeCostForBackups(SimplePath *p, int *potPathInd, int numPossiblePaths, int primaryInd, int *pathCosts, Channel cs[2*N_EDGES][MAX_CHANNELS]) {

    for(int i = 0; i < numPossiblePaths; ++i) {
        if(potPathInd[i] == -1) {
            pathCosts[i] = -1;
            break;
        }
        int pid = potPathInd[i];
        int cost = 0;

        for(int e = 0; e <= p[pid].index; ++e) {
            bool free = false;
            int edgeNum = (*p[pid].edges[e]).edgeNum;
            int firstOpenChannel = MAX_CHANNELS+1;

            for(int c = 0; !free && c < MAX_CHANNELS; ++c) {

                if(cs[edgeNum][c].primary == true) {
                    continue;
                }

                //At this point, we know that there are no primary paths on this channel
                //Thus we must check and see if it is "free".

                //we COULD use this channel, but there may be a "free" one further down.
                if(cs[edgeNum][c].numBackups == 0) {
                    if(c < firstOpenChannel) {
                        firstOpenChannel = c;
                    }
                    continue;
                }

                bool disjoint = true;

                //Check every connection currently on protected on the channel
                for(int bup = 0; disjoint && bup < channels[edgeNum][c].numBackups; ++bup) {

                    //At this point, we know that there is at least one path protected on this channel.
                    //Technically, we should also know that it's not a primary path.

                    //for each edge of the protected connection's primary path
                    for(int e2 = 0; disjoint && e2 <= (*(*channels[edgeNum][c].backupsOnChannel[bup]).primaryPath).index; ++e2) {

                        //see if its the same edge as used by our primary path.
                        for(int e3 = 0; disjoint && e3 <= p[primaryInd].index; ++e3 ) {

                            if((*(*channels[edgeNum][c].backupsOnChannel[bup]).primaryPath).edges[e2] == p[primaryInd].edges[e3]) {
                                //There is a non-disjoint primary path on this channel, so it is unusable.

                                disjoint = false;
                            }
                        }
                    }
                }

                if(disjoint) {
                    //This channel is free
                    free = true;
                }
            }

            if(!free) {
                if(firstOpenChannel < MAX_CHANNELS) {
                    cost++;
                }else {
                    cost = 1000000;
                    break;
                }

            }

        }

        pathCosts[i] = cost;
    }
}

//TODO: Give each thread an index into the array of simple paths, and have them check to see if "their" path is compatible.
int determineCompatibleBackups(SimplePath *p, int *potPathInd, int numPossiblePaths, int pInd) {
    int numDisjoint = 0;
    //First pass checks to see which simple paths are disjoint from the primary path.
    for(int i = 0; i < numPossiblePaths; ++i) {

        bool disjoint = true;
        //Check each edge to make sure they're disjoint
        for(int e1 = 0; disjoint && e1 <= p[pInd].index; ++e1) {
            for(int e2 = 0; disjoint && e2 <= p[i].index; ++e2) {
                if(p[i].edges[e2] == p[pInd].edges[e1]) {
                    disjoint = false;
                }
            }
        }
        if(disjoint) {
            potPathInd[numDisjoint] = i;
            numDisjoint++;
        }

    }
    //Mark the end of the array
    potPathInd[numDisjoint] = -1;
    //cout << "disjoint: " << numDisjoint << " out of " << numPossiblePaths <<"\n";
    return numDisjoint;
}

int computeAllSimplePathsN(SimplePath **ps, int *vertexList, Edge *edgeList, int sourceNode, int destNode, int hops) {
    int index = (sourceNode * N_NODES) + destNode;

    //initialize arrays
    int visited[N_NODES]; //visited[i] is 1 if node i has been visited on this path, 0 otherwise.
    int currentPath = 0;

    //edgeListIndex[i] contains the index into edgeList[] (aka the compact adj list) for node i.
    int edgeListIndex[N_NODES];

    ps[index][currentPath].index = 0;

    //Initialize our search components
    for(int i = 0; i < N_NODES; ++i) {
        visited[i] = 0;
        edgeListIndex[i] = vertexList[i];
    }

    stack <int> st;
    int currentNode;
    int neighbor;
    int currentHop = 1;

    st.push(sourceNode);
    visited[sourceNode] = 1;

    while(st.size() > 0) {
        //use loopCond to get to the beginning of the while loop from inside the for loop.
        bool loopCond = true;
        currentNode = st.top();
        //for each neighbor of currentNode
        for(; loopCond == true && edgeListIndex[currentNode] < vertexList[currentNode+1]; ++edgeListIndex[currentNode]) {
            neighbor = edgeList[edgeListIndex[currentNode]].v2;

            //If we're too far away from our source node, backtrack.
            if(currentHop >= hops) {
                break;
            }

            if(edgeList[edgeListIndex[currentNode]].load == MAX_CHANNELS) {
                continue;
            }

            //If our neighbor is the desired node, AND we're at the correct path length, save this path!
            if(neighbor == destNode && currentHop < hops) {

                ps[index][currentPath].edges[ps[index][currentPath].index] = &edgeList[edgeListIndex[currentNode]];

                ps[index][currentPath].sourceNode = sourceNode;
                ps[index][currentPath].destNode = destNode;
                ps[index][currentPath].hops = currentHop;

                //Copy the whole path up until the dest node to the next path in the array.
                //Note that we don't copy the COST from the current primary path, as the cost is computed
                //independently for each primary path.
                ps[index][currentPath+1].sourceNode = sourceNode;
                ps[index][currentPath+1].destNode = destNode;
                ps[index][currentPath+1].hops = currentHop;
                ps[index][currentPath+1].index = ps[index][currentPath].index-1;
                for(int i = 0; i < ps[index][currentPath].index; ++i) {
                    ps[index][currentPath+1].edges[i] = ps[index][currentPath].edges[i];
                }

                currentPath += 1;

                ps[index][currentPath].index += 1;
                ++edgeListIndex[currentNode];

                //
                loopCond = false;
                break;
            }

            if(!visited[neighbor]) {

                ps[index][currentPath].edges[ps[index][currentPath].index] = &edgeList[edgeListIndex[currentNode]];
                ps[index][currentPath].index += 1;

                st.push(neighbor);
                visited[neighbor] = 1;
                currentHop++;

                //continue the while loop, but increment the ELI first.
                ++edgeListIndex[currentNode];
                loopCond = false;
                break;
            }
        }

        if(loopCond) {
            currentHop--;

            //Once we've visited all of this node's neighbors, we reset it so that a
            //different path involving this node can be explored.
            visited[currentNode] = 0;
            ps[index][currentPath].index -= 1;

            edgeListIndex[currentNode] = vertexList[currentNode];
            st.pop();
        }

    }
    return currentPath;
}



void readGraphReorderEdgeList(int vertexList[],Edge compactEdgeList[2*N_EDGES],Edge reorderedEdgeList[2*N_NODES]) {
    //cout << "Beginning read\n";

    //TODO: We def don't need this extra array... please revise.
    int edgeList[N_NODES][N_NODES];
    for(int i = 0; i < N_NODES; ++i) {
        for(int j = 0; j < N_NODES; ++j) {
            edgeList[i][j] = 0;
        }
    }
    for(int i = 0; i < N_EDGES; ++i) {
        edgeList[base_edges[i][0]][base_edges[i][1]] = 1;
        edgeList[base_edges[i][1]][base_edges[i][0]] = 1;
    }

    int vDegree[N_NODES];

    int counter = 0;
    for(int i = 0; i < N_NODES; ++i) {
        vertexList[i] = counter;
        for(int j = 0; j < N_NODES; ++j) {
            if(edgeList[i][j] != 0) {
                compactEdgeList[counter].v1 = i;
                compactEdgeList[counter].v2 = j;
                compactEdgeList[counter].load = 0;
                compactEdgeList[counter].totalProtected = 0;
                compactEdgeList[counter].edgeNum = counter;

                //for(int x = 0; x < MAX_CHANNELS; ++x) {
                //    compactEdgeList[counter].channels[x].numBackups = 0;
                //}


                counter++;
            }
        }

        vDegree[i] = counter - vertexList[i];

        //cout << i << ": " << vDegree[i] << "\n";
    }
    vertexList[N_NODES] = 2*N_EDGES;

    //THis successfully reorders the edgelist based on the degree of the neighbor.
    //TODO: make this sorting algorithm faster... like WAY faster.
    for(int i = 0; i < N_NODES; ++i) {

        int startInd = vertexList[i];
        int endInd = vertexList[i+1];
        //[startInd,endInd)

        int reorderedInd = startInd;

        while(reorderedInd < endInd) {
            int min = startInd;
            int minVal = 66666; //min degree of the neighbor

            //Find the "smallest" neighbor of this node.
            for(int j = startInd; j < endInd; ++j) {

                bool isReordered = false;

                //Check to see if this node is already in our reordered list.
                for(int k = startInd; k < reorderedInd; ++k) {
                    if(reorderedEdgeList[k].v2 == compactEdgeList[j].v2) {
                        isReordered = true;
                        break;
                    }
                }

                //if its not in our reordered list and it qualifies as the minimum neighbor.
                if(isReordered == false && vDegree[compactEdgeList[j].v2] <= minVal) {
                    min = j;
                    minVal = vDegree[compactEdgeList[j].v2];
                }

            }

            reorderedEdgeList[reorderedInd].v1 = compactEdgeList[min].v1;
            reorderedEdgeList[reorderedInd].v2 = compactEdgeList[min].v2;
            reorderedEdgeList[reorderedInd].load = 0;
            reorderedInd++;
        }
    }
}
