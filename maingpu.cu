#include "hip/hip_runtime.h"
/*
 * File:   maingpu.cu
 * Author: jjbillings
 *
 * Created on October 16, 2016, 9:09 PM
 */

#include<cstdlib>
#include<stdio.h>
#include<queue>
#include<stack>
#include<iostream>
#include<fstream>
#include<ctime>
#include"nets.h"

using namespace std;

#define NUM_CONNECTIONS 500
#define MAX_CHANNELS 30
#define SAMPLES 1

struct SimplePath;
struct Path;
struct Edge;
struct Connection;
struct Connection;
struct Channel;

struct Channel{
    bool primary; //is this channel used for a primary path?
    int numBackups; //total protected;
    Connection *backupsOnChannel[NUM_CONNECTIONS];//Realistically, there will be far fewer than NUM_CONNECTIONS
    Connection *d_backupsOnChannel[NUM_CONNECTIONS];
};

struct Edge {
    int edgeNum;
    int v1;
    int v2;
    int load; //load <= MAX_CHANNELS. Also, load is the sum of the primary AND backups paths using it.
    int totalProtected;
};

struct SimplePath {
    int sourceNode;
    int destNode;

    int hops;
    int index;
    int edgeNums[N_NODES];

    Edge *edges[N_NODES];

  SimplePath() {
    for(int i = 0; i < N_NODES; ++i) {
      edgeNums[i] = -1;
      edges[i] = 0;
    }
    sourceNode = -1;
    destNode = -1;
    hops = -1;
    index = -1;
  };
};

struct Path {
    int sourceNode;
    int destNode;

    int hops;
    int index;
    int cost;

    //Every path that uses a particular edge just has a reference to it (not a copy), so they can each manipulate it.
    Edge *edges[N_NODES];
    bool freeEdges[N_NODES]; //whether or not that edge has a cost of 0
    int channelNum[N_NODES]; //Channel number for each edge that it uses
    int edgeNums[N_NODES];
    bool primary;
    bool active;
};

struct Connection {
    int sourceNode;
    int destNode;
    int combinedCost;
    bool validBackup;
    bool validPrimary;
    Path *backupPath;
    Path *primaryPath;
};

void readGraphReorderEdgeList(int vertexList[],Edge compactEdgeList[2*N_EDGES],Edge reorderedEdgeList[2*N_NODES]);

int computeAllSimplePathsN(SimplePath **ps, int *vertexList, Edge *edgeList, int sourceNode, int destNode, int hops);
void simulate(int *vertexList, Edge *edgeList);
void simulate_GPU(int *vertexList, Edge *edgeList);
void computeCostForBackupsWithGPU(SimplePath *p, int *potPathCosts, int primaryInd, Channel cs[2*N_EDGES][MAX_CHANNELS]);
int determineCompatibleBackups(SimplePath *p, int *potPathInd, int numPossiblePaths, int pInd);
void computeCostForBackups(SimplePath *p, int *potPathInd, int numPotPaths, int backupIndex, int *pathCosts,Channel cs[2*N_EDGES][MAX_CHANNELS]);
void selectChannels(Connection *c, Channel chan[2*N_EDGES][MAX_CHANNELS]);
void increaseLoad(Connection *connection, Channel channels[2*N_EDGES][MAX_CHANNELS], Connection *d_con);

int vertexList[N_NODES+1];
Edge edgeList[2*N_EDGES];
Edge reorderedEdgeList[2*N_EDGES];
Connection cons[NUM_CONNECTIONS];
Channel channels[2*N_EDGES][MAX_CHANNELS];



//-----------Kernel for Determining which Backups are compatible with which Primaries. WORKING---------//
__global__ void determineCompatibleBackups(SimplePath *ps, int *potPathCosts,int conInd){
 
  int p_ind = (conInd * NUM_CONNECTIONS) +  blockIdx.x;
  int b_ind = (conInd * NUM_CONNECTIONS) +  threadIdx.x;
  int output_ind = (blockIdx.x * NUM_CONNECTIONS) + threadIdx.x;

  int primIndex = ps[p_ind].index;
  int backIndex = ps[b_ind].index;

  int primHops = ps[p_ind].hops;
  int backHops = ps[b_ind].hops;
  
  if(primHops > 0 && backHops > 0) {
    bool disjoint = true;

    for(int e1 = 0; disjoint && e1 <= primIndex; ++e1) {
      for(int e2 = 0; disjoint && e2 <= backIndex; ++e2){
	if(ps[p_ind].edgeNums[e1] == ps[b_ind].edgeNums[e2]) {
	  disjoint = false;
	}
      }
    }
    if(disjoint) {
      potPathCosts[output_ind] = 1;
    }else {
      potPathCosts[output_ind] = -1;
    }
  }else {
    potPathCosts[output_ind] = -1;
  }
}

//-----------TEST_KERNEL_FOR_WARPS---------//
__global__ void determineCompatibleBackups2(SimplePath *ps, int *potPathCosts,int conInd){

  int warp_id = threadIdx.x / 32;
  int warp_offset = threadIdx.x % 32;
  
  int p_ind = (conInd * NUM_CONNECTIONS) +  blockIdx.x;
  int b_ind = (conInd * NUM_CONNECTIONS) +  threadIdx.x;
  int output_ind = (blockIdx.x * NUM_CONNECTIONS) + threadIdx.x;

  int primIndex = ps[p_ind].index;
  int backIndex = ps[b_ind].index;

  int primHops = ps[p_ind].hops;
  int backHops = ps[b_ind].hops;
  
  if(primHops > 0 && backHops > 0) {
    bool disjoint = true;

    for(int e1 = 0; disjoint && e1 <= primIndex; ++e1) {
      for(int e2 = 0; disjoint && e2 <= backIndex; ++e2){
	if(ps[p_ind].edgeNums[e1] == ps[b_ind].edgeNums[e2]) {
	  disjoint = false;
	}
      }
    }
    if(disjoint) {
      potPathCosts[output_ind] = 1;
    }else {
      potPathCosts[output_ind] = -1;
    }
  }else {
    potPathCosts[output_ind] = -1;
  }
}

__global__ void costsKernel(SimplePath *p, int *potPathCosts, int conInd , Channel *cs) {

  int p_ind = (conInd * NUM_CONNECTIONS) + blockIdx.x;
  int b_ind = (conInd * NUM_CONNECTIONS) + threadIdx.x;
  int index = (blockIdx.x * NUM_CONNECTIONS) + threadIdx.x;


  //If we already know that this combo is unusable, just quit.
        if(potPathCosts[index] == -1) {
	    return;
        }
        
        int cost = 0;

        for(int e = 0; e <= p[b_ind].index; ++e) {
            bool free = false;
            int edgeNum = p[b_ind].edgeNums[e];
	    
            int firstOpenChannel = MAX_CHANNELS+1;

            for(int c = 0; !free && c < MAX_CHANNELS; ++c) {

	        int channelIndex = (edgeNum * MAX_CHANNELS)+c;
	        if(cs[channelIndex].primary == true) {
                    continue;
                }

                //At this point, we know that there are no primary paths on this channel
                //Thus we must check and see if it is "free".

                //we COULD use this channel, but there may be a "free" one further down.
                if(cs[channelIndex].numBackups == 0) {
                    if(c < firstOpenChannel) {
                        firstOpenChannel = c;
                    }
                    continue;
                }

                bool disjoint = true;

                //Check every connection currently on protected on the channel
                for(int bup = 0; disjoint && bup < cs[channelIndex].numBackups; ++bup) {

                    //At this point, we know that there is at least one path protected on this channel.
                    //Technically, we should also know that it's not a primary path.

                    //for each edge of the protected connection's primary path
                    for(int e2 = 0; disjoint && e2 <= (*(*cs[channelIndex].d_backupsOnChannel[bup]).primaryPath).index; ++e2) {

                        //see if its the same edge as used by our primary path.
                        for(int e3 = 0; disjoint && e3 <= p[p_ind].index; ++e3 ) {

                            if((*(*cs[channelIndex].d_backupsOnChannel[bup]).primaryPath).edgeNums[e2] == p[p_ind].edgeNums[e3]) {
                                //There is a non-disjoint primary path on this channel, so it is unusable.

                                disjoint = false;
                            }
                        }
                    }
                }

                if(disjoint) {
                    //This channel is free
                    free = true;
                }
            }

            if(!free) {
                if(firstOpenChannel < MAX_CHANNELS) {
                    cost++;
                }else {
                    cost = 1000000;
                    break;
                }

            }

        }

        potPathCosts[index] = cost;
    
}

/*
 *TODO: I totally thought I made the algorithm be based on BFS, but it is in fact based on DFS.
 *So REVERSE the order of the edge list. Currently, the neighbor with the lowest degree gets pushed
 *to the "bottom" of the stack, so we end up computing the path with high-degree nodes in it...
 */
int main(int argc, char** argv) {
    cout <<"Welcome to main\n";

    hipFree(0);
    for(int f = 0; f < (2*N_EDGES); ++f){
        for(int g = 0; g < MAX_CHANNELS; ++g) {
            channels[f][g].numBackups = 0;
            channels[f][g].primary = false;
        }
    }
    readGraphReorderEdgeList(vertexList,edgeList,reorderedEdgeList);

    srand(time(NULL));

    simulate_GPU(vertexList,edgeList);
    //simulate(vertexList,edgeList);
    return 0;
}

void simulate_GPU(int *vertexList, Edge *edgeList){

    clock_t cpu_startTime, cpu_endTime;
    double cpu_elapsedTime = 0;
    float gpu_totalTime = 0;
    cpu_startTime = clock();
    
    int connectionNum = 0;
    const size_t sp_size = sizeof(SimplePath);
    const size_t potPathCosts_size = (NUM_CONNECTIONS * NUM_CONNECTIONS) * sizeof(int);
    const size_t ps_size = ((N_NODES*N_NODES)*NUM_CONNECTIONS)*sp_size; //Size of the entire 2D array
    const size_t row_size = NUM_CONNECTIONS*sp_size; //Size of a SINGLE row in the array of SimplePaths

    const size_t channels_size = ((2*N_EDGES)*MAX_CHANNELS)*sizeof(Channel);
    
    //Test Data
    int v1[40] = {9, 5, 6, 1, 3, 5, 4, 9, 9, 9, 7, 8, 2, 10, 3, 5, 9, 3, 2, 3, 5, 2, 3, 3, 10, 9, 10, 2, 1, 1, 3, 2, 9, 5, 4, 6, 10, 5, 0, 1};
    int v2[40] = {3, 8, 4, 3, 8, 3, 7, 1, 5, 6, 0, 6, 10, 5, 8, 2, 3, 6, 5, 4, 2, 3, 9, 7, 9, 5, 6, 5, 0, 2, 5, 5, 10, 3, 9, 3, 4, 1, 10, 2};
    
    SimplePath **ps = new SimplePath*[N_NODES * N_NODES]; //Storage for paths

    SimplePath *d_ps; //Device pointer for the array of SimplePaths
    int *d_potPathCosts; //Device pointer for the array of Potential Path Costs
    int *h_potPathCosts; //Host pointer for the array of potential path costs.

    Connection *d_cons; //Device pointer to the array of connections.
    Channel *d_channels; //Device pointer for the array of channels.
    
    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        ps[i] = new SimplePath[NUM_CONNECTIONS];
    }


    if(hipSuccess != hipMalloc((void **)&d_ps,ps_size)) {
    	cout << "Malloc Error\n";
    }else {
      cout << "allocated SimplePaths array on Device\n";
    }

    
    if(hipSuccess != hipMalloc((void **)&d_channels,channels_size)) {
	cout << "Error Allocating channels on GPU\n";
    }else {
	cout << "Allocated Channels array on GPU\n";
    }

    hipMalloc((void **)&d_cons,sizeof(Connection)*NUM_CONNECTIONS);
    

    hipMalloc((void **)&d_potPathCosts,potPathCosts_size);
    cout << "Allocated potential Path Costs array on device\n";

    hipMemcpy(d_channels,&channels,channels_size,hipMemcpyHostToDevice);

    
    h_potPathCosts = (int *)malloc(potPathCosts_size);

    //We COULD parallelize this by giving a thread a source/dest combo to compute the paths of. potentially beneficial for large graphs
    for(int src = 0; src < N_NODES; ++src) {
        for(int dest = 0; dest < N_NODES; ++dest) {
            if(src != dest) {
                int index = (src*N_NODES)+dest;
                computeAllSimplePathsN(ps,vertexList,edgeList,src,dest,N_NODES);
                //cout <<"All simple paths computed and stored! " << npaths[index] << " paths between " << src << " and " << dest << "\n";
            }
        }
    }


    //Copy Simple paths to the GPU
    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
      hipMemcpy(d_ps + (i*(NUM_CONNECTIONS)),ps[i],row_size,hipMemcpyHostToDevice);
    }

    //Setup components for GPU benchmarking.
    hipEvent_t start, stop;
    

    //cpu_startTime = clock();
    for(int c = 0; c < 5; ++c) {
      
      //Attempt to allocate SOME connection onto the network
      int s = v1[connectionNum];
      int d = v2[connectionNum];

    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Allocate storage for the potential primary/backup path combos
    int index = (s*N_NODES) + d;

    
    //BENCHMARKING
    hipEventRecord(start);
    
    //-----------Launch the Kernel-------------//
    determineCompatibleBackups<<<NUM_CONNECTIONS,NUM_CONNECTIONS>>>(d_ps, d_potPathCosts,index);
    //    hipDeviceSynchronize();

    //BENCHMARKING
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milli = 0;
    hipEventElapsedTime(&milli,start,stop);
    gpu_totalTime += milli;
    //cout << "Kernel Execution took: " << milli << " milliseconds\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    
    if(hipSuccess != hipGetLastError()) {
      cout << "CUDA ERROR IN KERNEL: " << hipGetLastError() << "\n";
    }

    //---------Launch the Kernel----------//
    costsKernel<<<NUM_CONNECTIONS,NUM_CONNECTIONS>>>(d_ps, d_potPathCosts, index,d_channels);
    
    //---------Copy the Results back to the host ---//
    hipMemcpy(h_potPathCosts,d_potPathCosts,potPathCosts_size,hipMemcpyDeviceToHost);    

    //for(int i = 0; i < NUM_CONNECTIONS; ++i) {
    //  computeCostForBackupsWithGPU(ps[index],h_potPathCosts,i,channels);
    //}

    //-----------Select the cheapest combo using GPU Results-----------//
    int minCostGPU = 100000000;
    int minPrimIndGPU = -1;
    int minBackIndGPU = -1;

    for(int p = 0; p < NUM_CONNECTIONS; ++p) {
        int primaryCostGPU = ps[index][p].hops;

        for(int b = 0; b < NUM_CONNECTIONS; ++b) {
	  int f = (p*NUM_CONNECTIONS)+b;
	  if(h_potPathCosts[(p*NUM_CONNECTIONS)+b] < 0) {
	      continue;
	  }
          if((h_potPathCosts[(p*NUM_CONNECTIONS)+b] + primaryCostGPU) < minCostGPU) {
                minCostGPU = (h_potPathCosts[(p*NUM_CONNECTIONS)+b] + primaryCostGPU);
                minPrimIndGPU = p;
                minBackIndGPU = b;
          }
        }
    }

    cout << "Min cost on GPU is: " << minCostGPU << "\n";
    cout << "PRIM: "<<minPrimIndGPU << "\n";
    for(int i = 0; i <= ps[index][minPrimIndGPU].index; ++i) {
      cout << (*ps[index][minPrimIndGPU].edges[i]).v1 << " -> " << (*ps[index][minPrimIndGPU].edges[i]).v2 << "\n";
    }
    cout << "BACK: " << minBackIndGPU << "\n";
    for(int i = 0; i <= ps[index][minBackIndGPU].index; ++i) {
      cout << (*ps[index][minBackIndGPU].edges[i]).v1 << " -> " << (*ps[index][minBackIndGPU].edges[i]).v2 << "\n";
    }


    //--------------Store the connection--------------//
    cons[connectionNum].sourceNode = s;
    cons[connectionNum].destNode = d;
    cons[connectionNum].combinedCost = minCostGPU;
    cons[connectionNum].validBackup = true;
    cons[connectionNum].validPrimary = true;
    cons[connectionNum].backupPath = new Path();
    cons[connectionNum].primaryPath = new Path();
    (*cons[connectionNum].primaryPath).hops = ps[index][minPrimIndGPU].hops;
    (*cons[connectionNum].primaryPath).index = ps[index][minPrimIndGPU].index;
    (*cons[connectionNum].primaryPath).primary = true;
    (*cons[connectionNum].backupPath).hops = ps[index][minBackIndGPU].hops;
    (*cons[connectionNum].backupPath).index = ps[index][minBackIndGPU].index;

    for(int p = 0; p <= ps[index][minPrimIndGPU].index; ++p) {
        (*cons[connectionNum].primaryPath).edges[p] = ps[index][minPrimIndGPU].edges[p];
        (*cons[connectionNum].primaryPath).freeEdges[p] = false;
	(*cons[connectionNum].primaryPath).edgeNums[p] = ps[index][minPrimIndGPU].edgeNums[p];
    }
    for(int p = 0; p <= ps[index][minBackIndGPU].index; ++p) {
        (*cons[connectionNum].backupPath).edges[p] = ps[index][minBackIndGPU].edges[p];
	(*cons[connectionNum].backupPath).edgeNums[p] = ps[index][minBackIndGPU].edgeNums[p];
    }

    

    //Select Channels
    selectChannels(&cons[connectionNum],channels);

    //Increase the network load
    increaseLoad(&cons[connectionNum],channels,&d_cons[connectionNum]);

    //NOTE: We can 100% only copy individual channels to the GPU. i.e. if only channels 3 and 41 were updated, we can copy ONLY those channels if we want to
    hipMemcpy(d_channels,&channels,channels_size,hipMemcpyHostToDevice);

    //TESTING FOR COSTS KERNEL
    hipMemcpy(d_cons,&cons,sizeof(Connection)*NUM_CONNECTIONS,hipMemcpyHostToDevice);
    
    //--------------Print Network Load--------------//
    for(int m = 0; m < 2*N_EDGES; ++m) {
        cout << "LOAD: " << edgeList[m].v1 << " -> " << edgeList[m].v2 << ": " << edgeList[m].load << " | TP: " << edgeList[m].totalProtected << " | ";
        if(edgeList[m].load > 0) {
            for(int c = 0; c < edgeList[m].load; ++c) {
                cout << "C" << c << ": " << channels[m][c].numBackups << " ";
                if(channels[m][c].primary == true) {
                    cout << "P ";
                }
            }
        }
        cout << "\n";

    }
    

    connectionNum++;
    }//ENDFOR

    hipDeviceSynchronize();
    cpu_endTime = clock();


    //--------------Clean up memory--------------//
    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        delete[] ps[i];
    }
    delete[] ps;
    
    hipFree(d_ps);
    hipFree(d_potPathCosts);
    hipFree(d_channels);
    hipFree(d_cons);
    
    free(h_potPathCosts);
    //cpu_endTime = clock();
    cpu_elapsedTime = ((double) (cpu_endTime - cpu_startTime)/CLOCKS_PER_SEC) * 1000;

        cout << "Kernel Execution took: " << gpu_totalTime << " milliseconds\n";
	cout << "Total time: " << cpu_elapsedTime << " milliseconds\n";
	//cout << "CPU Start: " << cpu_startTime << " CPU End: " << cpu_endTime << "\n";
}


void computeCostForBackupsWithGPU(SimplePath *p, int *potPathCosts, int primaryInd, Channel cs[2*N_EDGES][MAX_CHANNELS]) {

    for(int i = 0; i < NUM_CONNECTIONS; ++i) {
        int pid = (primaryInd * NUM_CONNECTIONS) + i;
        if(potPathCosts[pid] == -1) {
            continue;
        }
        
        int cost = 0;

        for(int e = 0; e <= p[i].index; ++e) {
            bool free = false;
            int edgeNum = (*p[i].edges[e]).edgeNum;
            int firstOpenChannel = MAX_CHANNELS+1;

            for(int c = 0; !free && c < MAX_CHANNELS; ++c) {

                if(cs[edgeNum][c].primary == true) {
                    continue;
                }

                //At this point, we know that there are no primary paths on this channel
                //Thus we must check and see if it is "free".

                //we COULD use this channel, but there may be a "free" one further down.
                if(cs[edgeNum][c].numBackups == 0) {
                    if(c < firstOpenChannel) {
                        firstOpenChannel = c;
                    }
                    continue;
                }

                bool disjoint = true;

                //Check every connection currently on protected on the channel
                for(int bup = 0; disjoint && bup < channels[edgeNum][c].numBackups; ++bup) {

                    //At this point, we know that there is at least one path protected on this channel.
                    //Technically, we should also know that it's not a primary path.

                    //for each edge of the protected connection's primary path
                    for(int e2 = 0; disjoint && e2 <= (*(*channels[edgeNum][c].backupsOnChannel[bup]).primaryPath).index; ++e2) {

                        //see if its the same edge as used by our primary path.
                        for(int e3 = 0; disjoint && e3 <= p[primaryInd].index; ++e3 ) {

                            if((*(*channels[edgeNum][c].backupsOnChannel[bup]).primaryPath).edges[e2] == p[primaryInd].edges[e3]) {
                                //There is a non-disjoint primary path on this channel, so it is unusable.

                                disjoint = false;
                            }
                        }
                    }
                }

                if(disjoint) {
                    //This channel is free
                    free = true;
                }
            }

            if(!free) {
                if(firstOpenChannel < MAX_CHANNELS) {
                    cost++;
                }else {
                    cost = 1000000;
                    break;
                }

            }

        }

        potPathCosts[pid] = cost;
    }
}








void simulate(int *vertexList, Edge *edgeList){
    clock_t cpu_startTime, cpu_endTime;
    double cpu_elapsedTime = 0;
    cpu_startTime = clock();

    //Test Data
    int v1[40] = {9, 5, 6, 1, 3, 5, 4, 9, 9, 9, 7, 8, 2, 10, 3, 5, 9, 3, 2, 3, 5, 2, 3, 3, 10, 9, 10, 2, 1, 1, 3, 2, 9, 5, 4, 6, 10, 5, 0, 1};
    int v2[40] = {3, 8, 4, 3, 8, 3, 7, 1, 5, 6, 0, 6, 10, 5, 8, 2, 3, 6, 5, 4, 2, 3, 9, 7, 9, 5, 6, 5, 0, 2, 5, 5, 10, 3, 9, 3, 4, 1, 10, 2};
    
    int connectionNum = 0;
    //We want to compute and store all possible paths between our source and desitination.
    SimplePath **ps = new SimplePath*[N_NODES * N_NODES]; //Storage for paths
    int *npaths = new int[N_NODES*N_NODES];

    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        ps[i] = new SimplePath[NUM_CONNECTIONS];
    }


    //We COULD parallelize this by giving a thread a source/dest combo to compute the paths of. potentially beneficial for large graphs
    for(int src = 0; src < N_NODES; ++src) {
        for(int dest = 0; dest < N_NODES; ++dest) {
            if(src != dest) {
                int index = (src*N_NODES)+dest;
                npaths[index] = computeAllSimplePathsN(ps,vertexList,edgeList,src,dest,N_NODES);
                //cout <<"All simple paths computed and stored! " << npaths[index] << " paths between " << src << " and " << dest << "\n";
            }
        }
    }
    //At this point, we COULD delete[] any paths in the array that we didn't use.


    for(int num = 0; num < 40; ++num) {
    //Attempt to allocate SOME connection onto the network
    int s = v1[connectionNum];
    int d = v2[connectionNum];
    //int s = rand() % N_NODES;
    //int d = rand() % N_NODES;
    //while(s == d) {
    //    s = rand()%N_NODES;
    //    d = rand()%N_NODES;
    //}

    //Allocate storage for the potential primary/backup path combos
    int index = (s*N_NODES) + d;
    int numPossiblePaths = npaths[index];

    //Stores indices into the ps[index][] array for each disjoint backup path.
    //potPathInd[i][j] = k where ps[index][k] is a path that is edge-disjoint from ps[index][i].
    int ** potPathInd = new int*[NUM_CONNECTIONS];
    for(int i = 0; i < NUM_CONNECTIONS; ++i) {
        potPathInd[i] = new int[NUM_CONNECTIONS];
    }


    //--------------Find all paths which are edge-disjoint from this primary--------------//
    int k = -1;
    //On the GPU, instead of iterating i..numPossiblePaths, we would give thread_i backup_i
    for(int i = 0; i < numPossiblePaths; ++i) {
        k = determineCompatibleBackups(ps[index],potPathInd[i],numPossiblePaths,i);
        //cout << "Number of paths which are disjoint from this primary path: " << k << "\n";
    }



    //--------------Compute Cost for each backup path--------------//
    int ** pathCosts = new int*[numPossiblePaths];
    for(int i = 0; i < numPossiblePaths; ++i) {
        pathCosts[i] = new int[numPossiblePaths];
    }

    for(int i = 0; i < numPossiblePaths; ++i) {
        computeCostForBackups(ps[index],potPathInd[i],numPossiblePaths,i,pathCosts[i],channels);
    }



    //--------------Select cheapest connection--------------//
    int minCost = 100000000;
    int minPrimInd = -1;
    int minBackInd = -1;

    for(int p = 0; p < numPossiblePaths; ++p) {
        int backInd = 0;
        int primaryCost = ps[index][p].hops;

        while(pathCosts[p][backInd] != -1) {
            if((pathCosts[p][backInd] + primaryCost) < minCost) {
                minCost = (pathCosts[p][backInd] + primaryCost);
                minPrimInd = p;
                minBackInd = backInd;
            }
            backInd++;
        }
    }
    cout << "Min cost is: " << minCost << "\n";



    //--------------Store the connection--------------//
    cons[connectionNum].sourceNode = s;
    cons[connectionNum].destNode = d;
    cons[connectionNum].combinedCost = minCost;
    cons[connectionNum].validBackup = true;
    cons[connectionNum].validPrimary = true;
    cons[connectionNum].backupPath = new Path();
    cons[connectionNum].primaryPath = new Path();
    (*cons[connectionNum].primaryPath).hops = ps[index][minPrimInd].hops;
    (*cons[connectionNum].primaryPath).index = ps[index][minPrimInd].index;
    (*cons[connectionNum].primaryPath).primary = true;
    (*cons[connectionNum].backupPath).hops = ps[index][potPathInd[minPrimInd][minBackInd]].hops;
    (*cons[connectionNum].backupPath).index = ps[index][potPathInd[minPrimInd][minBackInd]].index;

    for(int p = 0; p <= ps[index][minPrimInd].index; ++p) {
        (*cons[connectionNum].primaryPath).edges[p] = ps[index][minPrimInd].edges[p];
        (*cons[connectionNum].primaryPath).freeEdges[p] = false;
    }
    for(int p = 0; p <= ps[index][potPathInd[minPrimInd][minBackInd]].index; ++p) {
        (*cons[connectionNum].backupPath).edges[p] = ps[index][potPathInd[minPrimInd][minBackInd]].edges[p];
    }

    //Select Channels
    selectChannels(&cons[connectionNum],channels);

    //Increase the network load
    //    increaseLoad(&cons[connectionNum],channels); TODO: REIMPLEMENT WITH THE CORRECT NUM OF PARAMETERS FOR NON-GPU version.


    //--------------Print Network Load--------------//
    for(int m = 0; m < 2*N_EDGES; ++m) {
        cout << "LOAD: " << edgeList[m].v1 << " -> " << edgeList[m].v2 << ": " << edgeList[m].load << " | TP: " << edgeList[m].totalProtected << " | ";
        if(edgeList[m].load > 0) {
            for(int c = 0; c < edgeList[m].load; ++c) {
                cout << "C" << c << ": " << channels[m][c].numBackups << " ";
                if(channels[m][c].primary == true) {
                    cout << "P ";
                }
            }
        }
        cout << "\n";

    }


    //--------------Clean up memory--------------//
    for(int i = 0; i < numPossiblePaths; ++i) {
        delete[] potPathInd[i];
    }
    delete[] potPathInd;

    for(int i = 0; i < numPossiblePaths; ++i) {
        delete[] pathCosts[i];
    }
    delete[] pathCosts;
    connectionNum++;
}//end loop

    cpu_endTime = clock();
    
    for(int i = 0; i < (N_NODES*N_NODES); ++i) {
        delete[] ps[i];
    }
    delete[] ps;
    delete[] npaths;
    cout << "ps and npaths deleted\n";

    //cpu_endTime = clock();
    cpu_elapsedTime = ((double)(cpu_endTime-cpu_startTime)/CLOCKS_PER_SEC)*1000;
    cout << "CPU Total Elapsed Time: " << cpu_elapsedTime << "\n";
}

void increaseLoad(Connection *connection, Channel channels[2*N_EDGES][MAX_CHANNELS], Connection *d_con) {
    if((*(*connection).primaryPath).index < 0) {
        cout << "Primary Path DNE?\n";
        return;
    }
    //Increment the network load; put the backup on the channels

    //Here we are incrementing the network load for the PRIMARY PATH
    for(int i = 0; i <= (*(*connection).primaryPath).index; ++i) {

        //Every edge in the primary path gets its load increased
        channels[(*(*(*connection).primaryPath).edges[i]).edgeNum][(*(*connection).primaryPath).channelNum[i]].primary = true;
        channels[(*(*(*connection).primaryPath).edges[i]).edgeNum][(*(*connection).primaryPath).channelNum[i]].backupsOnChannel[0] = connection;
	channels[(*(*(*connection).primaryPath).edges[i]).edgeNum][(*(*connection).primaryPath).channelNum[i]].d_backupsOnChannel[0] = d_con;
        channels[(*(*(*connection).primaryPath).edges[i]).edgeNum][(*(*connection).primaryPath).channelNum[i]].numBackups += 1;
        (*(*(*connection).primaryPath).edges[i]).load += 1;
        (*(*(*connection).primaryPath).edges[i]).totalProtected += 1;
    }

    //Here we are increasing the network load for the BACKUP PATH
    for(int i = 0; i <= (*(*connection).backupPath).index; ++i) {
        //Temp
        Edge *e = (*(*connection).backupPath).edges[i];
        int cNum = (*(*connection).backupPath).channelNum[i];

        //first path to use this channel, or this is not a free edge for the backup path.
        //if(channels[(*e).edgeNum][cNum].numBackups == 0 || (*(*connection).backupPath).freeEdges[i] == false) {
        if((*(*connection).backupPath).freeEdges[i] == false) {
            (*e).load += 1;
        }

        //Marks that the connection is protected on this channel.
        int en = (*e).edgeNum;
        int numbs = channels[en][cNum].numBackups;
        channels[en][cNum].primary = false;
        channels[en][cNum].backupsOnChannel[numbs] = connection;
	channels[en][cNum].d_backupsOnChannel[numbs] = d_con;
        channels[en][cNum].numBackups += 1;
        (*e).totalProtected +=1;
    }

}

//TODO: This method contains a lot of redundant code that is also in computeCostForBackups. Consider combining.
//I wanted to modularize the code as much as possible this time around, which is why there's so much redundancy in this method.
void selectChannels(Connection *c, Channel chan[2*N_EDGES][MAX_CHANNELS]) {

  /*
    cout << "prim\n";
    for(int i = 0; i <= (*(*c).primaryPath).index; ++i) {
        cout << (*(*(*c).primaryPath).edges[i]).v1 << " -> " << (*(*(*c).primaryPath).edges[i]).v2 << "\n";
    }
    cout << "back\n";
    for(int i = 0; i <= (*(*c).backupPath).index; ++i) {
        cout << (*(*(*c).backupPath).edges[i]).v1 << " -> " << (*(*(*c).backupPath).edges[i]).v2 << "\n";
    }

  */
    int edgeNum = -1;
    //Select Primary path channels;
    for(int p = 0; p <= (*(*c).primaryPath).index; ++p){
        edgeNum = (*(*(*c).primaryPath).edges[p]).edgeNum;
        bool allSet = false;
        for(int ch = 0; !allSet && ch < MAX_CHANNELS; ++ch) {
            if(chan[edgeNum][ch].numBackups == 0) {
                allSet = true;
                (*(*c).primaryPath).channelNum[p] = ch;
            }
        }
    }

    for(int e = 0; e <= (*(*c).backupPath).index; ++e) {
        bool free = false;
        edgeNum = (*(*(*c).backupPath).edges[e]).edgeNum;
        int firstOpenChannel = MAX_CHANNELS+1;

        for(int ch = 0; !free && ch < MAX_CHANNELS; ++ch) {

            if(chan[edgeNum][ch].primary == true) {
                continue;
            }

            //At this point, we know that there are no primary paths on this channel
            //Thus we must check and see if it is "free".

            //we COULD use this channel, but there may be a "free" one further down.
            if(chan[edgeNum][ch].numBackups == 0) {
                if(ch < firstOpenChannel) {
                    firstOpenChannel = ch;
                }
                continue;
            }

            bool disjoint = true;

            //Check every connection currently on protected on the channel
            for(int bup = 0; disjoint && bup < chan[edgeNum][ch].numBackups; ++bup) {

                //At this point, we know that there is at least one path protected on this channel.
                //Technically, we should also know that it's not a primary path.

                //for each edge of the protected connection's primary path
                for(int e2 = 0; disjoint && e2 <= (*(*chan[edgeNum][ch].backupsOnChannel[bup]).primaryPath).index; ++e2) {

                    //see if its the same edge as used by our primary path.
                    for(int e3 = 0; disjoint && e3 <= (*(*c).primaryPath).index; ++e3 ) {

                        if((*(*chan[edgeNum][ch].backupsOnChannel[bup]).primaryPath).edges[e2] == (*(*c).primaryPath).edges[e3]) {
                            //There is a non-disjoint primary path on this channel, so it is unusable.
                            //goto CHANNEL_LOOP_END;
                            disjoint = false;
                        }
                    }
                }
            }

            if(disjoint) {
                //This channel is free
                free = true;
                (*(*c).backupPath).channelNum[e] = ch;
                (*(*c).backupPath).freeEdges[e] = true;
            }
        }

        if((*(*c).backupPath).freeEdges[e] == false) {
            (*(*c).backupPath).channelNum[e] = firstOpenChannel;
        }
    }
    //cout << "all set?\n";
}

void computeCostForBackups(SimplePath *p, int *potPathInd, int numPossiblePaths, int primaryInd, int *pathCosts, Channel cs[2*N_EDGES][MAX_CHANNELS]) {

    for(int i = 0; i < numPossiblePaths; ++i) {
        if(potPathInd[i] == -1) {
            pathCosts[i] = -1;
            break;
        }
        int pid = potPathInd[i];
        int cost = 0;

        for(int e = 0; e <= p[pid].index; ++e) {
            bool free = false;
            int edgeNum = (*p[pid].edges[e]).edgeNum;
            int firstOpenChannel = MAX_CHANNELS+1;

            for(int c = 0; !free && c < MAX_CHANNELS; ++c) {

                if(cs[edgeNum][c].primary == true) {
                    continue;
                }

                //At this point, we know that there are no primary paths on this channel
                //Thus we must check and see if it is "free".

                //we COULD use this channel, but there may be a "free" one further down.
                if(cs[edgeNum][c].numBackups == 0) {
                    if(c < firstOpenChannel) {
                        firstOpenChannel = c;
                    }
                    continue;
                }

                bool disjoint = true;

                //Check every connection currently on protected on the channel
                for(int bup = 0; disjoint && bup < channels[edgeNum][c].numBackups; ++bup) {

                    //At this point, we know that there is at least one path protected on this channel.
                    //Technically, we should also know that it's not a primary path.

                    //for each edge of the protected connection's primary path
                    for(int e2 = 0; disjoint && e2 <= (*(*channels[edgeNum][c].backupsOnChannel[bup]).primaryPath).index; ++e2) {

                        //see if its the same edge as used by our primary path.
                        for(int e3 = 0; disjoint && e3 <= p[primaryInd].index; ++e3 ) {

                            if((*(*channels[edgeNum][c].backupsOnChannel[bup]).primaryPath).edges[e2] == p[primaryInd].edges[e3]) {
                                //There is a non-disjoint primary path on this channel, so it is unusable.

                                disjoint = false;
                            }
                        }
                    }
                }

                if(disjoint) {
                    //This channel is free
                    free = true;
                }
            }

            if(!free) {
                if(firstOpenChannel < MAX_CHANNELS) {
                    cost++;
                }else {
                    cost = 1000000;
                    break;
                }

            }

        }

        pathCosts[i] = cost;
    }
}

//TODO: There's some sketchiness going on with numPossiblePaths vs NUM_CONNECTIONS.
int determineCompatibleBackups(SimplePath *p, int *potPathInd, int numPossiblePaths, int pInd) {
    int numDisjoint = 0;
    int numConf = 0;
    //cout << "SRC: " << p[pInd].sourceNode << " DEST: " << p[pInd].destNode << "\n";
    //First pass checks to see which simple paths are disjoint from the primary path.
    for(int i = 0; i < NUM_CONNECTIONS; ++i) {
      if(p[i].hops <= 0 || p[i].index < 0|| p[pInd].hops <= 0 || p[pInd].index < 0){numConf++; continue;}
        bool disjoint = true;
        //Check each edge to make sure they're disjoint
        for(int e1 = 0; disjoint && e1 <= p[pInd].index; ++e1) {
            for(int e2 = 0; disjoint && e2 <= p[i].index; ++e2) {
                if(p[i].edgeNums[e2] == p[pInd].edgeNums[e1]) {
                    disjoint = false;
		    numConf++;
                }
            }
        }
        if(disjoint) {
            potPathInd[numDisjoint] = i;
            numDisjoint++;
        }

    }
    //Mark the end of the array
    potPathInd[numDisjoint] = -1;
    //cout << "disjoint: " << numDisjoint << " out of " << numPossiblePaths <<"\n";
    //cout << "conflicts: " << numConf << "\n";
    return numDisjoint;
}

int computeAllSimplePathsN(SimplePath **ps, int *vertexList, Edge *edgeList, int sourceNode, int destNode, int hops) {
    int index = (sourceNode * N_NODES) + destNode;

    //initialize arrays
    int visited[N_NODES]; //visited[i] is 1 if node i has been visited on this path, 0 otherwise.
    int currentPath = 0;

    //edgeListIndex[i] contains the index into edgeList[] (aka the compact adj list) for node i.
    int edgeListIndex[N_NODES];

    ps[index][currentPath].index = 0;

    //Initialize our search components
    for(int i = 0; i < N_NODES; ++i) {
        visited[i] = 0;
        edgeListIndex[i] = vertexList[i];
    }

    stack <int> st;
    int currentNode;
    int neighbor;
    int currentHop = 1;

    st.push(sourceNode);
    visited[sourceNode] = 1;

    while(st.size() > 0) {
        //use loopCond to get to the beginning of the while loop from inside the for loop.
        bool loopCond = true;
        currentNode = st.top();
        //for each neighbor of currentNode
        for(; loopCond == true && edgeListIndex[currentNode] < vertexList[currentNode+1]; ++edgeListIndex[currentNode]) {
            neighbor = edgeList[edgeListIndex[currentNode]].v2;

            //If we're too far away from our source node, backtrack.
            if(currentHop >= hops) {
                break;
            }

            if(edgeList[edgeListIndex[currentNode]].load == MAX_CHANNELS) {
                continue;
            }

            //If our neighbor is the desired node, AND we're at the correct path length, save this path!
            if(neighbor == destNode && currentHop < hops) {

                ps[index][currentPath].edges[ps[index][currentPath].index] = &edgeList[edgeListIndex[currentNode]];
		ps[index][currentPath].edgeNums[ps[index][currentPath].index] = edgeList[edgeListIndex[currentNode]].edgeNum;
		
                ps[index][currentPath].sourceNode = sourceNode;
                ps[index][currentPath].destNode = destNode;
                ps[index][currentPath].hops = currentHop;

                //Copy the whole path up until the dest node to the next path in the array.
                //Note that we don't copy the COST from the current primary path, as the cost is computed
                //independently for each primary path.
                ps[index][currentPath+1].sourceNode = sourceNode;
                ps[index][currentPath+1].destNode = destNode;
                ps[index][currentPath+1].hops = currentHop;
                ps[index][currentPath+1].index = ps[index][currentPath].index-1;
                for(int i = 0; i < ps[index][currentPath].index; ++i) {
                    ps[index][currentPath+1].edges[i] = ps[index][currentPath].edges[i];
		    ps[index][currentPath+1].edgeNums[i] = ps[index][currentPath].edgeNums[i];
                }

                currentPath += 1;

                ps[index][currentPath].index += 1;
                ++edgeListIndex[currentNode];

                //
                loopCond = false;
                break;
            }

            if(!visited[neighbor]) {

                ps[index][currentPath].edges[ps[index][currentPath].index] = &edgeList[edgeListIndex[currentNode]];
		ps[index][currentPath].edgeNums[ps[index][currentPath].index] = edgeList[edgeListIndex[currentNode]].edgeNum;
                ps[index][currentPath].index += 1;

                st.push(neighbor);
                visited[neighbor] = 1;
                currentHop++;

                //continue the while loop, but increment the ELI first.
                ++edgeListIndex[currentNode];
                loopCond = false;
                break;
            }
        }

        if(loopCond) {
            currentHop--;

            //Once we've visited all of this node's neighbors, we reset it so that a
            //different path involving this node can be explored.
            visited[currentNode] = 0;
            ps[index][currentPath].index -= 1;

            edgeListIndex[currentNode] = vertexList[currentNode];
            st.pop();
        }

    }
    //Last path is invalid
    ps[index][currentPath].hops = 0;

    for(int j = currentPath; j < NUM_CONNECTIONS; ++j) {
      ps[index][j].hops = 0;
    }
    return currentPath;
}



void readGraphReorderEdgeList(int vertexList[],Edge compactEdgeList[2*N_EDGES],Edge reorderedEdgeList[2*N_NODES]) {
    //cout << "Beginning read\n";

    //TODO: We def don't need this extra array... please revise.
    int edgeList[N_NODES][N_NODES];
    for(int i = 0; i < N_NODES; ++i) {
        for(int j = 0; j < N_NODES; ++j) {
            edgeList[i][j] = 0;
        }
    }
    for(int i = 0; i < N_EDGES; ++i) {
        edgeList[base_edges[i][0]][base_edges[i][1]] = 1;
        edgeList[base_edges[i][1]][base_edges[i][0]] = 1;
    }

    int vDegree[N_NODES];

    int counter = 0;
    for(int i = 0; i < N_NODES; ++i) {
        vertexList[i] = counter;
        for(int j = 0; j < N_NODES; ++j) {
            if(edgeList[i][j] != 0) {
                compactEdgeList[counter].v1 = i;
                compactEdgeList[counter].v2 = j;
                compactEdgeList[counter].load = 0;
                compactEdgeList[counter].totalProtected = 0;
                compactEdgeList[counter].edgeNum = counter;

                //for(int x = 0; x < MAX_CHANNELS; ++x) {
                //    compactEdgeList[counter].channels[x].numBackups = 0;
                //}


                counter++;
            }
        }

        vDegree[i] = counter - vertexList[i];

        //cout << i << ": " << vDegree[i] << "\n";
    }
    vertexList[N_NODES] = 2*N_EDGES;

    //THis successfully reorders the edgelist based on the degree of the neighbor.
    //TODO: make this sorting algorithm faster... like WAY faster.
    for(int i = 0; i < N_NODES; ++i) {

        int startInd = vertexList[i];
        int endInd = vertexList[i+1];
        //[startInd,endInd)

        int reorderedInd = startInd;

        while(reorderedInd < endInd) {
            int min = startInd;
            int minVal = 66666; //min degree of the neighbor

            //Find the "smallest" neighbor of this node.
            for(int j = startInd; j < endInd; ++j) {

                bool isReordered = false;

                //Check to see if this node is already in our reordered list.
                for(int k = startInd; k < reorderedInd; ++k) {
                    if(reorderedEdgeList[k].v2 == compactEdgeList[j].v2) {
                        isReordered = true;
                        break;
                    }
                }

                //if its not in our reordered list and it qualifies as the minimum neighbor.
                if(isReordered == false && vDegree[compactEdgeList[j].v2] <= minVal) {
                    min = j;
                    minVal = vDegree[compactEdgeList[j].v2];
                }

            }

            reorderedEdgeList[reorderedInd].v1 = compactEdgeList[min].v1;
            reorderedEdgeList[reorderedInd].v2 = compactEdgeList[min].v2;
            reorderedEdgeList[reorderedInd].load = 0;
            reorderedInd++;
        }
    }
}
